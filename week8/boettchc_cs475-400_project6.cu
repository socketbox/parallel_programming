#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <cstring>
#include <sstream>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

#ifndef BLOCKSIZE
#define BLOCKSIZE		32		// number of threads per block
#endif

// setting the number of trials in the monte carlo simulation:
#ifndef NUMTRIALS
#define NUMTRIALS  1000000
#endif

// how many tries to discover the maximum performance:
#ifndef NUMTRIES
#define NUMTRIES  10
#endif

// ranges for the random numbers (changed for proj6):
const float XCMIN =   0.0;
const float XCMAX =   2.0;
const float YCMIN =   0.0;
const float YCMAX =   2.0;
const float RMIN  =   0.5;
const float RMAX  =   2.0;

// function prototypes:
float Ranf( float, float );
int Ranf( int, int );
void TimeOfDaySeed( );
bool does_file_exist(const char *name);
void write_results_to_csv_file(char *prog_name, int threads, int trials, float mega_trials);


// laser tag  (CUDA Kernel) on the device
__global__  void ShootLaser( float *A, float *B, float *C, int *D )
{
	
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

	// randomize the location and radius of the circle:
	float xc = A[gid];
	float yc = B[gid];
	float  r =  C[gid];

	/*chb: DEBUG:
	printf("This is xc in kernel: %lf\n", xc);
	printf("This is yc in kernel: %lf\n", yc);
	printf("This is r in kernel: %lf\n", r);*/

	// solve for the intersection using the quadratic formula:
	float a = 2.;
	float b = -2.*( xc + yc );
	float c = xc*xc + yc*yc - r*r;
	float d = b*b - 4.*a*c;
	
	//If d is less than 0, then the circle was completely missed. (Case A) 
	//Continue on to the next trial in the for-loop.
	if(d < 0)
		D[gid] = 0;
  else
	{
		// hits the circle:
		// get the first intersection:
		d = sqrt( d );
		float t1 = (-b + d ) / ( 2.*a );  // time to intersect the circle
		float t2 = (-b - d ) / ( 2.*a );  // time to intersect the circle
		float tmin = t1 < t2 ? t1 : t2;    // only care about the first intersection

		//If tmin is less than 0., then the circle completely engulfs the laser pointer. (Case B) Continue on to the next 
		//trial in the for-loop.
		if(tmin < 0)
			D[gid] = 0;
		else
		{
			// where does it intersect the circle?
			float xcir = tmin;
			float ycir = tmin;

			// get the unitized normal vector at the point of intersection:
			float nx = xcir - xc;
			float ny = ycir - yc;
			float x = sqrt( nx*nx + ny*ny );
			nx /= x;  // unit vector
			ny /= x;  // unit vector

			// get the unitized incoming vector:
			float inx = xcir - 0.;
			float iny = ycir - 0.;
			float in = sqrt( inx*inx + iny*iny );
			inx /= in;  // unit vector
			iny /= in;  // unit vector

			// get the outgoing (bounced) vector:
			float dot = inx*nx + iny*ny;
			//float outx = inx - 2.*nx*dot;  // angle of reflection = angle of incidence`
			float outy = iny - 2.*ny*dot;  // angle of reflection = angle of incidence`

			// find out if it hits the infinite plate:
			float t = ( 0. - ycir ) / outy;
			
			//If t is less than 0., then the reflected beam went up instead of down. 
			//Continue on to the next trial in the for-loop.
			if(t < 0)
				D[gid] = 0;
			else
			{
				D[gid] = 1;
			}
		}
	}
}

// main program:
int main( int argc, char* argv[ ] )
{
  TimeOfDaySeed( ); // seed the random number generator
	
	//int dev = findCudaDevice(argc, (const char **)argv);
	
	// allocate host memory for...
  // xc	
	float * hA = new float [ NUMTRIALS ];
	// yc	
	float * hB = new float [ NUMTRIALS ];
	// radius	
	float * hC = new float [ NUMTRIALS ];
	// hit/miss	
	int * hD = new int[ NUMTRIALS ];

	// fill the random-value arrays:
	for( int n = 0; n < NUMTRIALS; n++ )
	{       
		hA[n] = Ranf( XCMIN, XCMAX );
		hB[n] = Ranf( YCMIN, YCMAX );
		hC[n] = Ranf(  RMIN,  RMAX ); 
		hD[n] = 0;
	}      


	// allocate device memory:
	float *dA, *dB, *dC; 
	int	*dD;

	dim3 dimsA( NUMTRIALS, 1, 1 );
	dim3 dimsB( NUMTRIALS, 1, 1 );
	dim3 dimsC( NUMTRIALS, 1, 1 );
	dim3 dimsD( NUMTRIALS, 1, 1 );

	hipError_t status;
	status = hipMalloc( reinterpret_cast<void **>(&dA), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dB), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dC), (NUMTRIALS)*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dD), (NUMTRIALS)*sizeof(int) );
		checkCudaErrors( status );

	// copy host memory to the device:
	status = hipMemcpy( dA, hA, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dB, hB, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dC, hC, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );

	// setup the execution parameters:
	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( NUMTRIALS / threads.x, 1, 1 );

	// Create and start timer
	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:
	hipEvent_t start, stop;
	status = hipEventCreate( &start );
		checkCudaErrors( status );
	status = hipEventCreate( &stop );
		checkCudaErrors( status );

	// get ready to record the maximum performance and the probability:
	float maxPerformance = 0.;      
	float currentProb = 0.;              
	float maxCurrentProb = 0.;           
	double megaTrialsPerSecond = 0.;
	float msecTotal = 0.;
	double secondsTotal = 0.;
	int numHits = 0;
	
		// looking for the maximum performance:
	for( int t = 0; t < NUMTRIES; t++ )
	{
		// record the start event:
		status = hipEventRecord( start, NULL );
		checkCudaErrors( status );
		
		// execute the kernel:
		ShootLaser<<< grid, threads >>>( dA, dB, dC, dD );
	
		//record the stop event:
		status = hipEventRecord( stop, NULL );
		checkCudaErrors( status );

		// wait for the stop event to complete:
		status = hipEventSynchronize( stop );
			checkCudaErrors( status );

		status = hipEventElapsedTime( &msecTotal, start, stop );
		checkCudaErrors( status );

		// compute and print the performance
		secondsTotal = 0.001 * (double)msecTotal;
		//double multsPerSecond = (float)SIZE * (float)NUMTRIALS / secondsTotal;
		//double megaMultsPerSecond = multsPerSecond / 1000000.;

		//copy result from the device to the host:
		status = hipMemcpy( hD, dD, (NUMTRIALS)*sizeof(int), hipMemcpyDeviceToHost );
		checkCudaErrors( status );

		numHits = 0;
		// check the sum :
		for(int i = 0; i < NUMTRIALS; i++ )
		{
			if(hD[i] == 1)
			numHits++;
		}
		megaTrialsPerSecond = (double)NUMTRIALS / secondsTotal / 1000000.;
		if( megaTrialsPerSecond > maxPerformance )
			maxPerformance = megaTrialsPerSecond;
		currentProb = (float)numHits/(float)NUMTRIALS;
		if(currentProb > maxCurrentProb)
			maxCurrentProb = currentProb;
	}
	
	printf("Block Size: %i\tTrials: %i\tHit Probability: %lf\tTime Delta: %lf\tMegaTrials/sec: %lf\n",\
			BLOCKSIZE, NUMTRIALS, maxCurrentProb, secondsTotal, maxPerformance);

	write_results_to_csv_file(argv[0], BLOCKSIZE, NUMTRIALS, maxPerformance); 

	// clean up memory:
	delete [ ] hA;
	delete [ ] hB;
	delete [ ] hC;
	delete [ ] hD;

	status = hipFree( dA );
		checkCudaErrors( status );
	status = hipFree( dB );
		checkCudaErrors( status );
	status = hipFree( dC );
		checkCudaErrors( status );
	status = hipFree( dD );
		checkCudaErrors( status );

	return 0;
		
}

float Ranf( float low, float high )
{
        float r = (float) rand();               // 0 - RAND_MAX
        float t = r  /  (float) RAND_MAX;       // 0. - 1.

        return   low  +  t * ( high - low );
}

int Ranf( int ilow, int ihigh )
{
        float low = (float)ilow;
        float high = ceil( (float)ihigh );

        return (int) Ranf(low,high);
}

void TimeOfDaySeed( )
{
  struct tm y2k = { 0 };
  y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
  y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

  time_t  timer;
  time( &timer );
  double seconds = difftime( timer, mktime(&y2k) );
  unsigned int seed = (unsigned int)( 1000.*seconds );    // milliseconds
  srand( seed );
}

bool does_file_exist(const char *name)
{
    if (FILE *file = fopen(name, "r")) 
		{
        fclose(file);
        return true;
    } 
		else 
        return false;
}

void write_results_to_csv_file(char *prog_name, int blocksize, int numtrials, float perf)
{
	//program name + .csv extension + null terminator
	char ext[5] = ".csv";	
	char *filename = std::strcat(prog_name, ext);

	std::ostringstream converter;
	//build strings for writing a line at a time
	std::string results;
	converter << blocksize  << "," << numtrials << "," << perf << std::endl;
	results = converter.str();	
	//std::string results_mflops(std::to_string(threads) + "," + \
      //std::to_string(trials) + "," + std::to_string(hit_prob) + "," + std::to_string(mega_trials));

  std::ofstream outFile_mflops;
  
  if (!does_file_exist(filename))
	{
    outFile_mflops.open(filename, std::ios::out);
		std::string header("blocksize,numtrials,perf");
		outFile_mflops << header << std::endl;
	}
	else
    outFile_mflops.open(filename, std::ios::app);
  
  outFile_mflops << results << std::endl;
  //file closed via RAII
}


